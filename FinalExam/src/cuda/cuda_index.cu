#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"
#include "iostream"
#include "chrono"
#include "time.h"

#include "../utils/matrix/matrix.h"

using namespace std;
using namespace std::chrono;

// Number of threads to execute the Matrix Multiplication
int num_of_blocks;
int threads_per_block;

// Total execution time expressed in milliseconds
double total_execution_time_ms = 0.0;

__global__ void mul_matrixes(long *a, long *b, long *result, int *dimensions)
{
  for (int y = blockIdx.x; y < *dimensions; y+=gridDim.x)
  {
    for (int x = threadIdx.x; x < *dimensions; x+=blockDim.x)
    {
      long current = 0;
      for (int i = 0; i < *dimensions; i++)
      {
        int a_index = (y * (*dimensions)) + i;
        int b_index = (i * (*dimensions)) + x;

        current += ((*(a + a_index)) * (*(b + b_index)));
      }
      int r_index = (y * (*dimensions)) + x;
      *(result + r_index) = current;
    }
  }
}

void setup_mul_matrixes(struct Matrix *a, struct Matrix *b, struct Matrix *result)
{
  hipError_t cuda_err = hipSuccess;
  int memory_to_allocate = sizeof(long) * result->area;

  // --------------------------------------------------------------------- //
  // Data setup in device memory
  long *device_a, *device_b, *device_result;
  int *dimensions;

  // Allocate memory in device
  cuda_err = hipMalloc((void**)&device_a, memory_to_allocate);
  cuda_err = hipMalloc((void**)&device_b, memory_to_allocate);
  cuda_err = hipMalloc((void**)&device_result, memory_to_allocate);
  cuda_err = hipMalloc((void**)&dimensions, sizeof(int));

  // Copy the data to the device memory
  cuda_err = hipMemcpy(device_a, a->pointer, memory_to_allocate, hipMemcpyHostToDevice);
  cuda_err = hipMemcpy(device_b, b->pointer, memory_to_allocate, hipMemcpyHostToDevice);
  cuda_err = hipMemcpy(dimensions, &(b->dimensions), sizeof(int), hipMemcpyHostToDevice);

  // --------------------------------------------------------------------- //
  // Lauch the kernel
  mul_matrixes<<<num_of_blocks, threads_per_block>>>(device_a, device_b, device_result, dimensions);

  // Verify the execution
  cuda_err = hipGetLastError();
  if (cuda_err)
  {
    perror("Failed to launch distort face kernel");
    exit(EXIT_FAILURE);
  }
  hipDeviceSynchronize();

  // --------------------------------------------------------------------- //
  // Copy the results into the main memory
  cuda_err = hipMemcpy(result->pointer, device_result, memory_to_allocate, hipMemcpyDeviceToHost);

  // Free the device memory
  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_result);
  hipFree(dimensions);
}

int main(int argc, const char **argv)
{
  // Take the start time
  auto start_time = high_resolution_clock::now();

  // Check the amoung of arguments received
  if (argc != 4)
  {
    cout << "Invalid number of arguments" << endl;
    exit(-1);
  }

  // Fill the variables with the values received via arguments
  num_of_blocks = atoi(*(argv + 1));
  threads_per_block = atoi(*(argv + 2));
  int matrix_size = atoi(*(argv + 3));

  // Create the matrixes
  {
    time_t nTime;
    int seed = (int) time(&nTime);
    setup_seed(seed);
  }
  struct Matrix *matrix_a = generate_matrix(matrix_size, true);
  struct Matrix *matrix_b = generate_matrix(matrix_size, true);
  struct Matrix *matrix_result = generate_matrix(matrix_size, false);

  setup_mul_matrixes(matrix_a, matrix_b, matrix_result);

  // cout << "Matrix A:" << endl;
  // print_matrix(matrix_a);
  // cout << endl;

  // cout << "Matrix B:" << endl;
  // print_matrix(matrix_b);
  // cout << endl;

  // cout << "Result:" << endl;
  // print_matrix(matrix_result);
  // cout << endl;

  free_matrix(matrix_a);
  free_matrix(matrix_b);
  free_matrix(matrix_result);

  // Take the finish time
  auto finish_time = high_resolution_clock::now();
  // Calculate the algorithm duration time
  auto duration = duration_cast<microseconds>(finish_time - start_time);
  total_execution_time_ms = duration.count();

  cout
  << "The matrixes were multiplied in "
  << total_execution_time_ms / 1e6
  << " seconds"
  << endl;

  return 0;
}
