#include "hip/hip_runtime.h"
#include "opencv4/opencv2/objdetect.hpp"
#include "opencv4/opencv2/highgui.hpp"
#include "opencv4/opencv2/imgproc.hpp"
#include "opencv4/opencv2/videoio.hpp"
#include "opencv4/opencv2/core/utility.hpp"
#include "opencv4/opencv2/opencv.hpp"

using namespace std;
using namespace cv;

// Used to distord the face
int convolution_matrix_size;
int pixels_in_convolution_matrix;

// Num of block and threads per block to use in the GPU
int threads_per_block;

// Blocks of the image than will be processed on each MP
int extended_factor;
int image_block_width = 8, image_block_height = 8, image_block_area = 64;
int extended_image_block_width, extended_image_block_height, extended_image_block_area;

void setup_filter(int convolution_matrix_size_p, int threads_per_block_p)
{
  convolution_matrix_size = convolution_matrix_size_p;
  pixels_in_convolution_matrix = convolution_matrix_size_p * convolution_matrix_size_p;
  extended_factor = convolution_matrix_size / 2;

  // --------------------------------------------------------------------- //
  threads_per_block = threads_per_block_p;

  // --------------------------------------------------------------------- //
  extended_image_block_width = image_block_width + convolution_matrix_size - 1;
  extended_image_block_height = image_block_height + convolution_matrix_size - 1;
  extended_image_block_area = extended_image_block_width * extended_image_block_height;
}

// Available local memory 24 bytes
// Available shared memory
__global__ void distort_face(short *r, short *g, short *b, int *additional_data)
{
  __shared__ int
    blocks_per_row,
    pixels_in_convolution_matrix,
    extended_factor,
    image_block_width,
    image_block_height,
    extended_image_width;

  if (threadIdx.x == 0)
  {
    blocks_per_row = *(additional_data);
    pixels_in_convolution_matrix = *(additional_data + 1);
    extended_factor = *(additional_data + 2);
    image_block_width = *(additional_data + 6);
    image_block_height = *(additional_data + 7);
    extended_image_width = *(additional_data + 12);
  }
  __syncthreads();

  int x = ((blockIdx.x % blocks_per_row) * image_block_width) + (threadIdx.x % image_block_width);
  int y = ((blockIdx.x / blocks_per_row) * image_block_height) + (threadIdx.x / image_block_height);

  int reg_r = 0, reg_g = 0, reg_b = 0, index = 0;

  for (int reg_y = y + extended_factor * 2; reg_y >= y; reg_y--)
  {
    for (int reg_x = x + extended_factor * 2; reg_x >= x; reg_x--)
    {
      index = reg_y * extended_image_width + reg_x;
      reg_r += *(r + index);
      reg_g += *(g + index);
      reg_b += *(b + index);
    }
  }

  x += extended_factor;
  y += extended_factor;
  index = y * extended_image_width + x;

  *(r + index) = reg_r / pixels_in_convolution_matrix;
  *(g + index) = reg_g / pixels_in_convolution_matrix;
  *(b + index) = reg_b / pixels_in_convolution_matrix;
}

// Get the data from the Frame and copy it into the r, g, b arrays
void mat_to_pointers(Mat frame, Rect extended_face, short *r, short *g, short *b)
{
  int pointer = 0;
  for (int y = 0; y < extended_face.height; y++)
  {
    for (int x = 0; x < extended_face.width; x++)
    {
      Vec3b pixel = frame.at<Vec3b>(y + extended_face.y, x + extended_face.x);
      *(r + pointer) = pixel[0];
      *(g + pointer) = pixel[1];
      *(b + pointer) = pixel[2];
      pointer++;
    }
  }
}

// Get the data from the r, g, b arrays and copy it into the Frame
void pointers_to_mat(Mat frame, Rect extended_face, short *r, short *g, short *b)
{
  int pointer = 0;
  for (int y = 0; y < extended_face.height; y++)
  {
    for (int x = 0; x < extended_face.width; x++)
    {
      Vec3b &pixel = frame.at<Vec3b>(y + extended_face.y, x + extended_face.x);
      pixel[0] = *(r + pointer);
      pixel[1] = *(g + pointer);
      pixel[2] = *(b + pointer);
      pointer++;
    }
  }
}

void apply_filter(Mat frame, Rect face)
{
  hipError_t cuda_err = hipSuccess;

  // Calculate the face dimenssions according to detected and returned in face
  Rect fixed_face = Rect(
    face.x,
    face.y,
    face.width + (image_block_width - (face.width % image_block_width)),
    face.height + (image_block_height - (face.height % image_block_height))
  );
     
  // Calculate the extended face dimensions having into accound the pixels
  // required to apply the convolution matrix
  Rect extended_face = Rect(
    fixed_face.x - extended_factor,
    fixed_face.y - extended_factor,
    fixed_face.width + convolution_matrix_size - 1,
    fixed_face.height + convolution_matrix_size - 1
  );

  int blocks_per_row = fixed_face.width / image_block_width;

  // Size of memory in bytes to allocate in both device and main memmories
  int memory_to_allocate = extended_face.area() * sizeof(short);
  int additional_data_to_allocate = sizeof(int) * 15;

  // --------------------------------------------------------------------- //
  // Data setup in main memory
  short *r, *g, *b;
  int *additional_data;

  r = (short *)malloc(memory_to_allocate);
  g = (short *)malloc(memory_to_allocate);
  b = (short *)malloc(memory_to_allocate);
  additional_data = (int *)malloc(additional_data_to_allocate);

  // Check the result of malloc call
  if (r == NULL || g == NULL || b == NULL || additional_data == NULL)
  {
    perror("Error allocationg memory for rbg in the main memory");
    exit(-1);
  }

  // Copy the data from the Frame into the r, g, b arrays
  mat_to_pointers(frame, extended_face, r, g, b);

  // Copy the data into Block dimensions
  *additional_data = blocks_per_row;
  *(additional_data + 1) = pixels_in_convolution_matrix;
  *(additional_data + 2) = extended_factor;

  *(additional_data + 3) = extended_image_block_width;
  *(additional_data + 4) = extended_image_block_height;
  *(additional_data + 5) = extended_image_block_area;

  *(additional_data + 6) = image_block_width;
  *(additional_data + 7) = image_block_height;
  *(additional_data + 8) = image_block_area;

  *(additional_data + 9) = fixed_face.width;
  *(additional_data + 10) = fixed_face.height;
  *(additional_data + 11) = fixed_face.area();

  *(additional_data + 12) = extended_face.width;
  *(additional_data + 13) = extended_face.height;
  *(additional_data + 14) = extended_face.area();

  // --------------------------------------------------------------------- //
  // Data setup in device memory
  short *device_r, *device_g, *device_b;
  int *device_additional_data;

  cuda_err = hipMalloc((void**)&device_r, memory_to_allocate);
  cuda_err = hipMalloc((void**)&device_g, memory_to_allocate);
  cuda_err = hipMalloc((void**)&device_b, memory_to_allocate);
  cuda_err = hipMalloc((void**)&device_additional_data, additional_data_to_allocate);

  // Copy the data to the device memory
  cuda_err = hipMemcpy(device_r, r, memory_to_allocate, hipMemcpyHostToDevice);
  cuda_err = hipMemcpy(device_g, g, memory_to_allocate, hipMemcpyHostToDevice);
  cuda_err = hipMemcpy(device_b, b, memory_to_allocate, hipMemcpyHostToDevice);
  cuda_err = hipMemcpy(device_additional_data, additional_data, additional_data_to_allocate, hipMemcpyHostToDevice);

  // --------------------------------------------------------------------- //
  // Lauch the kernel (Apply the filter)
  distort_face<<<fixed_face.area() / image_block_area, threads_per_block>>>(
    device_r,
    device_g,
    device_b,
    device_additional_data
  );

  // Verify the execution
  cuda_err = hipGetLastError();
  if (cuda_err)
  {
    perror("Failed to launch distort face kernel");
    exit(EXIT_FAILURE);
  }
  hipDeviceSynchronize();

  // --------------------------------------------------------------------- //
  // Copy the results into the main memory
  hipMemcpy(r, device_r, memory_to_allocate, hipMemcpyDeviceToHost);
  hipMemcpy(g, device_g, memory_to_allocate, hipMemcpyDeviceToHost);
  hipMemcpy(b, device_b, memory_to_allocate, hipMemcpyDeviceToHost);
  hipMemcpy(additional_data, device_additional_data, additional_data_to_allocate, hipMemcpyDeviceToHost);

  // Free the device memory
  hipFree(device_r);
  hipFree(device_g);
  hipFree(device_b);
  hipFree(device_additional_data);

  // --------------------------------------------------------------------- //
  // Copy the results into the frame
  pointers_to_mat(frame, extended_face, r, g, b);

  // Free the main memory
  free(r);
  free(g);
  free(b);
  free(additional_data);
}